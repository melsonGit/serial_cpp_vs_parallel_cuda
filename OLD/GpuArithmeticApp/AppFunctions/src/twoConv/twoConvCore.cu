#include "hip/hip_runtime.h"
#include "../../inc/twoConv/twoConvCore.cuh"

using Clock = std::chrono::steady_clock;

void twoConvCore()
{
    // Initialise and allocate variable conSize with a user selected value
	int conSize { twoConvConSet(conSize) };

    // Initialise and allocate native arrays (hostMainVec; input) and (hostResultVec; output) a container size of conSize * conSize
    int* hostMainVec { new int[conSize * conSize] };
    int* hostResultVec { new int[conSize * conSize] };

    // Initialise and allocate the mask a container size of maskDim * maskDim
    int* hostMaskVec { new int[maskAttributes::maskDim * maskAttributes::maskDim] };

    // Populate input arrays
    std::cout << "\n2D Convolution: Populating main vector.\n";
    twoConvNumGen(hostMainVec, conSize);
    std::cout << "\n2D Convolution: Populating mask vector.\n";
    twoConvNumGen(hostMaskVec, maskAttributes::maskDim);

    std::cout << "\n2D Convolution: Populating complete.\n";

    // Initialise bytesVecMem/MaskMem to used for allocating memory to device vars
    // This allows us to copy data host to device and vice versa.
    size_t bytesVecMem { conSize * conSize * sizeof(int) };
    size_t bytesMaskMem { maskAttributes::maskDim * maskAttributes::maskDim * sizeof(int) };

    // Allocate memory on the device using hipMalloc
    int* deviceMainVec, * deviceMaskVec, * deviceResultVec;
    hipMalloc(&deviceMainVec, bytesVecMem);
    hipMalloc(&deviceMaskVec, bytesMaskMem);
    hipMalloc(&deviceResultVec, bytesVecMem);

    std::cout << "\n2D Convolution: Copying data from host to device.\n";

    // Copy data from the host to the device using hipMemcpy | .data() returns pointer to memory used by vector/array to store its owned elements
    hipMemcpy(deviceMainVec, hostMainVec, bytesVecMem, hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskVec, hostMaskVec, bytesMaskMem, hipMemcpyHostToDevice);

    // Threads per Cooperative Thread Array
    int THREADS { 32 };

    // No. CTAs per grid
    // Add padding | Enables compatibility with sample sizes not divisible by 32
    int BLOCKS { (conSize + THREADS - 1) / THREADS };

    // Use dim3 structs for BLOCKS and THREADS dimensions | Passed to kernal lauch as launch arguments
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    std::cout << "\n2D Convolution: Starting operation.\n";

    // Start clock
    auto opStart { Clock::now() };

    // Launch kernel on device
    twoConvFunc <<< blocks, threads >>> (deviceMainVec, deviceMaskVec, deviceResultVec, conSize);

    // Stop clock
    auto opEnd { Clock::now() };

    std::cout << "\n2D Convolution: Operation complete.\n";
    std::cout << "\n2D Convolution: Copying results from device to host.\n";

    // Copy data from device back to host using hipMemcpy
    hipMemcpy(hostResultVec, deviceResultVec, bytesVecMem, hipMemcpyDeviceToHost);

    std::cout << "\n2D Convolution: Copying complete.\n";

    // Authenticate results on host
    twoConvCheck(hostMainVec, hostMaskVec, hostResultVec, conSize);
    
    std::cout << "\n2D Convolution: Freeing host and device memory.\n\n";

    // Free allocated memory on the host and device
    delete[] hostMainVec;
    delete[] hostResultVec;
    delete[] hostMaskVec;

    hipFree(deviceMainVec);
    hipFree(deviceResultVec);

    // Output timing to complete operation and container size
    std::cout << "GPU 2D Convolution computation time (container size: " << conSize * conSize << "):\n"
              << std::chrono::duration_cast<std::chrono::microseconds>(opEnd - opStart).count() << " us\n"
              << std::chrono::duration_cast<std::chrono::milliseconds>(opEnd - opStart).count() << " ms\n\n"
              << "Returning to selection screen.\n\n"

              << "#########################################################################\n" <<
                 "#########################################################################\n" <<
                 "#########################################################################\n\n";
}