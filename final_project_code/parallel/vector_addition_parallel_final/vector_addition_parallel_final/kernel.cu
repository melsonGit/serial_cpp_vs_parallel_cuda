#include "hip/hip_runtime.h"
﻿// Parallel Vector Addition Program
/*
 Code sourced and adpated from the following author/s and sourcess:
- https://solarianprogrammer.com/2012/04/11/vector-addition-benchmark-c-cpp-fortran/
- https://github.com/CoffeeBeforeArch/cuda_programming/blob/master/vectorAdd/baseline/vectorAdd.cu
- https://www.youtube.com/watch?v=QVVTsLmMlwk&t
- https://thispointer.com/how-to-fill-a-vector-with-random-numbers-in-c/
 Please refer to the bibliography for a complete reference of the above author/s and sources
*/

#include <algorithm>
#include <iostream>
#include <vector>

using std::cout;
using std::cin;
using std::generate;
using std::vector;

// Function Prototypes
int element_set(int);

// Kernal that will be called from host (CPU) and run on the device (GPU)
// Function that computes the sum of two arrays
// CUDA kernel for vector addition || __global__ means this is called from the CPU, and runs on the GPU
__global__ void vectorAdd(const int* __restrict a, const int* __restrict b,
    int* __restrict c, int no_elements) {
    // Calculate global thread ID
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Boundary check
    if (tid < no_elements) c[tid] = a[tid] + b[tid];

}

int main() {

    // Call element_set function to assign variable no_elements with a user selected value
    static int no_elements = element_set(no_elements);
    size_t bytes = sizeof(int) * no_elements;

    // Vectors for holding the host-side (CPU-side) data
    vector<int> a;
    vector<int> b;
    vector<int> c;

    a.reserve(no_elements);
    b.reserve(no_elements);
    c.reserve(no_elements);

    // Initialise vector by generating random numbers via Lambda C++11 function
    generate(a.begin(), a.end(), []() {
        return rand() % 100;
        });
    generate(b.begin(), b.end(), []() {
        return rand() % 100;
        });

    // Alternative but slower || Initialize random numbers in each array
    //for (int i = 0; i < no_elements; i++) {
    //    a.push_back(rand() % 100);
    //    b.push_back(rand() % 100);
    //}

    // Allocate memory on the device (GPU)
    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Start the clock
    clock_t start = clock();

    // Copy data from the host to the device (CPU -> GPU)
    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

    // Threads per Cooperative Thread Array (CTA; 1024)
    int NUM_THREADS = 1 << 10;

    // CTAs per Grid
    // We need to launch at LEAST as many threads as we have elements
    // This equation pads an extra CTA to the grid if N cannot evenly be divided
    // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
    int NUM_BLOCKS = (no_elements + NUM_THREADS - 1) / NUM_THREADS;

    // Launch the kernel on the GPU
    // Kernel calls are asynchronous (the CPU program continues execution after
    // call, but not necessarily before the kernel finishes)
    vectorAdd << <NUM_BLOCKS, NUM_THREADS >> > (d_a, d_b, d_c, no_elements);

    // Copy sum vector from device to host
    // hipMemcpy is a synchronous operation, and waits for the prior kernel
    // launch to complete (both go to the default stream in this case).
    // Therefore, this hipMemcpy acts as both a memcpy and synchronization
    // barrier.
    hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    // Stop the clock just after the vectorAdd function finishes executing
    clock_t end = clock();

    double diffs = (end - start) / (double)CLOCKS_PER_SEC;
    cout << diffs << "s Vector Addition computation time, with an element size of " << no_elements << ".\n";
    cout << "PARALLEL VECTOR ADDITION COMPUTATION SUCCESSFUL.\nShutting down program....\n";

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return EXIT_SUCCESS;
}

// Function Declarations
int element_set(int element_size) {

    int temp_input;

    cout << "Please select vector addition element sample size from the options below:\n";
    cout << "1. 55,000,000\n";
    cout << "2. 100,000,000\n";
    cout << "3. 150,000,000\n";
    cout << "4. 200,000,000\n";
    cout << "5. 280,000,000\n";
    cin >> temp_input;

    if (temp_input <= 0 || temp_input >= 6)
    {
        cout << "\n\nNo correct option selected!\nShutting down program....\n";
        return EXIT_FAILURE;
    }

    if (temp_input == 1) {
        element_size = 55000000;
    }
    else if (temp_input == 2) {
        element_size = 100000000;
    }
    else if (temp_input == 3) {
        element_size = 150000000;
    }
    else if (temp_input == 4) {
        element_size = 200000000;
    }
    else if (temp_input == 5) {
        element_size = 280000000;
    }

    return element_size;
}
