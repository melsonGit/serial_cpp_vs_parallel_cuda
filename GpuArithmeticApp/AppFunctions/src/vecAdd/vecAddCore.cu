#include "hip/hip_runtime.h"
#include "../../inc/vecAdd/vecAddCore.cuh"

using Clock = std::chrono::steady_clock;

void vecAddCore()
{
	// Initialise and allocate variable conSize with a user selected value
	int conSize { vecAddConSet(conSize) };

	// Initialise and allocate input vectors (hostInputVecA & hostInputVecB) and output vector (hostResultVec) a container size of conSize
	std::vector<int> hostInputVecA(conSize), hostInputVecB(conSize), hostResultVec(conSize);

	// Populate input vectors
	std::cout << "\nVector Addition: Populating 1 of 2 input vectors.\n";
	vecAddNumGen(hostInputVecA);
	std::cout << "\nVector Addition: Populating 2 of 2 input vectors.\n";
	vecAddNumGen(hostInputVecB);

	std::cout << "\nVector Addition: Populating complete.\n";

	// Initialise bytesVecMem to used for allocating memory to device vars
	// This allows us to copy data host to device and vice versa.
	size_t bytesVecMem { sizeof(int) * conSize };

	// Allocate memory on the device using hipMalloc
	int* deviceInputVecA, * deviceInputVecB, * deviceResultVec;
	hipMalloc(&deviceInputVecA, bytesVecMem);
	hipMalloc(&deviceInputVecB, bytesVecMem);
	hipMalloc(&deviceResultVec, bytesVecMem);

	std::cout << "\nVector Addition: Copying data from host to device.\n";

	// Copy data from the host to the device using hipMemcpy | .data() returns pointer to memory used by vector/array to store its owned elements
	hipMemcpy(deviceInputVecA, hostInputVecA.data(), bytesVecMem, hipMemcpyHostToDevice);
	hipMemcpy(deviceInputVecB, hostInputVecB.data(), bytesVecMem, hipMemcpyHostToDevice);

	// Threads per Cooperative Thread Array
	int THREADS { 32 };

	// No. CTAs per grid
	// Add padding | Enables compatibility with sample sizes not divisible by 32
	int BLOCKS { (conSize + THREADS - 1) / THREADS };

	std::cout << "\nVector Addition: Starting operation.\n";

	// Start clock
	auto opStart { Clock::now() };

	// Launch kernel on device
	vecAddFunc << <BLOCKS, THREADS >> > (deviceInputVecA, deviceInputVecB, deviceResultVec, conSize);

	// Stop clock
	auto opEnd { Clock::now() };

	std::cout << "\nVector Addition: Operation complete.\n";
	std::cout << "\nVector Addition: Copying results from device to host.\n";

	// Copy data from device back to host using hipMemcpy
	hipMemcpy(hostResultVec.data(), deviceResultVec, bytesVecMem, hipMemcpyDeviceToHost);

	std::cout << "\nVector Addition: Copying complete.\n";

	// Authenticate results on host
	vecAddCheck(hostInputVecA, hostInputVecB, hostResultVec, conSize);

	std::cout << "\nVector Addition: Freeing device memory.\n\n";

	// Free memory on device
	hipFree(deviceInputVecA);
	hipFree(deviceInputVecB);
	hipFree(deviceResultVec);

	// Output timing to complete operation and container size
	std::cout << "GPU Vector Addition computation time (container size: " << conSize << "):\n"
			  << std::chrono::duration_cast<std::chrono::microseconds>(opEnd - opStart).count() << " us\n"
			  << std::chrono::duration_cast<std::chrono::milliseconds>(opEnd - opStart).count() << " ms\n\n"
			  << "Returning to selection screen.\n\n"

			  << "#########################################################################\n" <<
				 "#########################################################################\n" <<
				 "#########################################################################\n\n";
}