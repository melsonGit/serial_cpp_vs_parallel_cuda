#include "hip/hip_runtime.h"
#include "../../inc/matMulti/matMultiCore.cuh"

void matMultiCore()
{
	// Initialise and allocate variable conSize with a user selected value
	int conSize { matMultiConSet(conSize) };

	// Initialise and allocate native host input vectors (hostInputVecA & hostInputVecB) and the native host output vector (hostResultVec) a container size of conSize * conSize
	std::vector<int> hostInputVecA(conSize * conSize), hostInputVecB(conSize * conSize), hostResultVec(conSize * conSize);
	
	// Populate input vectors
	std::cout << "\nMatrix Multiplication: Populating 1 of 2 host input vectors.\n";
	matMultiNumGen(hostInputVecA);
	std::cout << "\nMatrix Multiplication: Populating 2 of 2 host input vectors.\n";
	matMultiNumGen(hostInputVecB);

	std::cout << "\nMatrix Multiplication: Populating complete.\n";

	// Initialise bytesVecMem to used for allocating memory to device vars
	// This allows us to copy data host to device and vice versa.
	size_t bytesVecMem { conSize * conSize * sizeof(int) };

	// Allocate memory on the device using hipMalloc
	int* deviceInputVecA, * deviceInputVecB, * deviceResultVec;
	hipMalloc(&deviceInputVecA, bytesVecMem);
	hipMalloc(&deviceInputVecB, bytesVecMem);
	hipMalloc(&deviceResultVec, bytesVecMem);

	std::cout << "\nMatrix Multiplication: Copying data from host to device.\n";

	// Copy data from the host to the device using hipMemcpy | .data() returns pointer to memory used by vector/array to store its owned elements
	hipMemcpy(deviceInputVecA, hostInputVecA.data(), bytesVecMem, hipMemcpyHostToDevice);
	hipMemcpy(deviceInputVecB, hostInputVecB.data(), bytesVecMem, hipMemcpyHostToDevice);

	// Threads per Cooperative Thread Array
	int THREADS { 32 };

	// No. CTAs per grid
	// Add padding | Enables compatibility with sample sizes not divisible by 32
	int BLOCKS { (conSize + THREADS - 1) / THREADS };

	// Use dim3 structs for BLOCKS and THREADS dimensions | Passed to kernal lauch as launch arguments
	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	// Start the clock
	clock_t opStart { clock() };

	std::cout << "\nMatrix Multiplication: Starting operation.\n";

	// Launch kernel on device
	matMultiFunc<<<blocks, threads>>>(deviceInputVecA, deviceInputVecB, deviceResultVec, conSize);

	std::cout << "\nMatrix Multiplication: Operation complete.\n";

	// Stop clock
	clock_t opEnd { clock() };

	std::cout << "\nMatrix Multiplication: Copying results from device to host.\n";

	// Copy data from device back to host using hipMemcpy
	hipMemcpy(hostResultVec.data(), deviceResultVec, bytesVecMem, hipMemcpyDeviceToHost);

	std::cout << "\nMatrix Multiplication: Copying complete.\n";

	// Authenticate results on host
	matMultiCheck(hostInputVecA, hostInputVecB, hostResultVec, conSize);

	std::cout << "\nMatrix Multiplication: Freeing device memory.\n\n";

	// Free memory on device
	hipFree(deviceInputVecA);
	hipFree(deviceInputVecB);
	hipFree(deviceResultVec);

	// Calculate overall time spent to complete operation
	double completionTime { ((static_cast<double>(opEnd)) - (static_cast<double>(opStart))) / (double)CLOCKS_PER_SEC };

	// Output timing to complete operation and container size
	std::cout << completionTime << "s Matrix Multiplication computation time, with a container size of " << conSize * conSize << ".\n\n";
	std::cout << "Returning to selection screen.\n\n";

	std::cout << "#########################################################################\n" <<
				 "#########################################################################\n" <<
			     "#########################################################################\n\n";
}