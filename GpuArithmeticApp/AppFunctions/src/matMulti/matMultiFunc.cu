#include "hip/hip_runtime.h"
#include "../../inc/matMulti/matMultiFunc.cuh"

__global__ void matMultiFunc(const int* inputVecA, const int* inputVecB, int* resultVec, const int conSize)
{
    // Compute each thread's global row and column index
    int rowId = blockIdx.y * blockDim.y + threadIdx.y;
    int colId = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over row, and down column
    resultVec[rowId * conSize + colId] = 0;

    for (auto rowColPairId { 0 }; rowColPairId < conSize; ++rowColPairId)
    {
        // Accumulate results for a single element
        resultVec[rowId * conSize + colId] += inputVecA[rowId * conSize + rowColPairId] * inputVecB[rowColPairId * conSize + colId];
    }
}