#include "hip/hip_runtime.h"
#include "../../inc/oneConv/oneConvCore.cuh"

// Number of elements in the convolution mask
constexpr int maskOneDim { 7 };

void oneConvCore()
{
	// Assign variable conSize with a user selected value
	int conSize { oneConvConSet(conSize) };

	// Size of the vector in bytes
	size_t bytesVecMem { conSize * sizeof(int) };
	// Size of mask in bytes
	size_t bytesMaskMem { maskOneDim * sizeof(int) };

	// Allocate main vector and resultant vector with size conSize
	std::vector<int> hostMainVec(conSize), hostResVec(conSize);
	// Allocate mask vector with MASK_ONE_DIM
	std::vector<int> hostMaskVec(maskOneDim);

	// Popluate main vector and mask vector
	std::cout << "\n1D Convolution: Populating main vector.\n";
	oneConvNumGen(hostMainVec);
	std::cout << "\n1D Convolution: Populating mask vector.\n";
	oneConvNumGen(hostMaskVec);

	std::cout << "\n1D Convolution: Populating complete.\n";

	// Allocate space on the device
	int* deviceMainVec, * deviceMaskVec, * deviceResVec;
	hipMalloc(&deviceMainVec, bytesVecMem);
	hipMalloc(&deviceMaskVec, bytesMaskMem);
	hipMalloc(&deviceResVec, bytesVecMem);

	// Copy host input vector data to the device input pointers
	std::cout << "\n1D Convolution: Copying data from host to device.\n";

	// Copy the data to the device
	hipMemcpy(deviceMainVec, hostMainVec.data(), bytesVecMem, hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskVec, hostMaskVec.data(), bytesMaskMem, hipMemcpyHostToDevice);

	// Threads per TB (thread blocks)
	int THREADS = 256;

	// Number of TBs with padding
	int BLOCKS{ (conSize + THREADS - 1) / THREADS };

	// Start clock
	clock_t opStart{ clock() };

	// Launch kernel
	std::cout << "\n1D Convolution: Starting operation.\n";

	// Call the kernel
	oneConvFunc << <BLOCKS, THREADS >> > (deviceMainVec, deviceMaskVec, deviceResVec, conSize, maskOneDim);

	std::cout << "\n1D Convolution: Operation complete.\n";

	// Stop clock
	clock_t opEnd{ clock() };

	std::cout << "\n1D Convolution: Copying results from device to host.\n";

	// Copy back to the host
	hipMemcpy(hostResVec.data(), deviceResVec, bytesVecMem, hipMemcpyDeviceToHost);

	oneConvCheck(hostMainVec.data(), hostMaskVec.data(), hostResVec.data(), conSize, maskOneDim);

	std::cout << "\n1D Convolution: Freeing device memory.\n\n";

	// Free allocated memory on the device and host
	hipFree(deviceResVec);
	hipFree(deviceMaskVec);
	hipFree(deviceMainVec);

	// Calculate overall time spent to complete operation
	double completionTime{ ((static_cast<double>(opEnd)) - (static_cast<double>(opStart))) / (double)CLOCKS_PER_SEC };

	// Output timing to complete operation and container size
	std::cout << completionTime << "s 1D Convolution computation time, with a container size of " << conSize << ".\n\n";
	std::cout << "Returning to selection screen.\n\n";

	std::cout << "#########################################################################\n" <<
				 "#########################################################################\n" <<
				 "#########################################################################\n\n";
}