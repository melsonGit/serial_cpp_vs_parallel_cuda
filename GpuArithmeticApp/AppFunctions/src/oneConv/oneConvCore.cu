#include "hip/hip_runtime.h"
#include "../../inc/oneConv/oneConvCore.cuh"

void oneConvCore()
{
	// Initialise and allocate variable conSize with a user selected value
	int conSize { oneConvConSet(conSize) };

	// Initialise and allocate main vector and resultant vector with size conSize
	std::vector<int> hostMainVec(conSize), hostResVec(conSize);

	// Initialise and allocate mask vector with maskDim
	std::vector<int> hostMaskVec(maskAttributes::maskDim);

	// Popluate input vectors
	std::cout << "\n1D Convolution: Populating main vector.\n";
	oneConvNumGen(hostMainVec);
	std::cout << "\n1D Convolution: Populating mask vector.\n";
	oneConvNumGen(hostMaskVec);

	std::cout << "\n1D Convolution: Populating complete.\n";

	// Initialise bytesVecMem/MaskMem to used for allocating memory to device vars
	// This allows us to copy data host to device and vice versa.
	size_t bytesVecMem { conSize * sizeof(int) };
	size_t bytesMaskMem { maskAttributes::maskDim * sizeof(int) };

	// Allocate memory on the device using hipMalloc
	int* deviceMainVec, * deviceMaskVec, * deviceResVec;
	hipMalloc(&deviceMainVec, bytesVecMem);
	hipMalloc(&deviceMaskVec, bytesMaskMem);
	hipMalloc(&deviceResVec, bytesVecMem);

	std::cout << "\n1D Convolution: Copying data from host to device.\n";

	// Copy data from the host to the device using hipMemcpy | .data() returns pointer to memory used by vector/array to store its owned elements
	hipMemcpy(deviceMainVec, hostMainVec.data(), bytesVecMem, hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskVec, hostMaskVec.data(), bytesMaskMem, hipMemcpyHostToDevice);

	// Threads per Cooperative Thread Array
	int THREADS { 32 };

	// No. CTAs per grid
	// Add padding | Enables compatibility with sample sizes not divisible by 32
	int BLOCKS { (conSize + THREADS - 1) / THREADS };

	// Start the clock
	clock_t opStart { clock() };

	std::cout << "\n1D Convolution: Starting operation.\n";

	// Launch kernel on device
	oneConvFunc << <BLOCKS, THREADS >> > (deviceMainVec, deviceMaskVec, deviceResVec, conSize);

	std::cout << "\n1D Convolution: Operation complete.\n";

	// Stop clock
	clock_t opEnd { clock() };

	std::cout << "\n1D Convolution: Copying results from device to host.\n";

	// Copy data from device back to host using hipMemcpy
	hipMemcpy(hostResVec.data(), deviceResVec, bytesVecMem, hipMemcpyDeviceToHost);

	std::cout << "\n2D Convolution: Copying complete.\n";

	// Authenticate results on host
	oneConvCheck(hostMainVec.data(), hostMaskVec.data(), hostResVec.data(), conSize);

	std::cout << "\n1D Convolution: Freeing device memory.\n\n";

	// Free allocated memory on the device
	hipFree(deviceResVec);
	hipFree(deviceMaskVec);
	hipFree(deviceMainVec);

	// Calculate overall time spent to complete operation
	double completionTime { ((static_cast<double>(opEnd)) - (static_cast<double>(opStart))) / (double)CLOCKS_PER_SEC };

	// Output timing to complete operation and container size
	std::cout << completionTime << "s 1D Convolution computation time, with a container size of " << conSize << ".\n\n";
	std::cout << "Returning to selection screen.\n\n";

	std::cout << "#########################################################################\n" <<
				 "#########################################################################\n" <<
				 "#########################################################################\n\n";
}