#include "hip/hip_runtime.h"
#include "../../inc/oneConv/oneConvFunc.cuh"

__global__ void oneConvFunc(const int* mainVec, const int* maskVec, int* resVec, const int conSize) 
{
    // Global thread ID calculation
    int rowId { blockIdx.x * blockDim.x + threadIdx.x };

    // maskRadius will determine when convolution occurs to prevent out of bound errors
    int maskRadius { MASK_ONE_DIM / 2 };

    // Calculate the starting point for the element
    int startPoint { rowId - maskRadius };

    // Go over each element of the mask
    for (auto j { 0 }; j < MASK_ONE_DIM; j++)
    {
        // Ignore elements that hang off (0s don't contribute)
        if (((startPoint + j) >= 0) && (startPoint + j < conSize)) 
        {
            // Collate results
            resVec[rowId] += mainVec[startPoint + j] * maskVec[j];
        }
    }
}